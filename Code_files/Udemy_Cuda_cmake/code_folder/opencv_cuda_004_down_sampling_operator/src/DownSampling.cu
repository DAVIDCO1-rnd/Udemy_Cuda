#include "hip/hip_runtime.h"
// Nvcc predefines the macro __HIPCC__.
// This macro can be used in sources to test whether they are currently being compiled by nvcc.
#ifndef __HIPCC__
#error Must be compiled with CUDA compiler!
#endif

#include <stdio.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "ConvertFromInteger.cuh"
#include "CudaMain.cuh"
#include "DownSampling.cuh"
#include "DownSampling_curnel.cu"


// Support DS for segentation (NO billinear)

__wchar_t* DownSampleTopLeft(
	void* deviceInputBuffer, void* deviceOutputBuffer,
	int widthSourceImage, int heightSourceImage, int strideSourceImage,
	int widthDestImage, int heightDestImage, int strideDestImage,
	double horizontalScale, double verticalScale,
	int subPixelType, int maxValue, int alphaChannelNumber, int pixelSize, int channelSize,
	int blockSizeX, int blockSizeY, int blockSizeZ, int gridSizeX, int gridSizeY)
{
	return DownSampleTopLeft_Parallel(deviceInputBuffer, deviceOutputBuffer, widthSourceImage, heightSourceImage,
		strideSourceImage, widthDestImage, heightDestImage, strideDestImage, horizontalScale, verticalScale,
		subPixelType, maxValue, alphaChannelNumber, pixelSize, channelSize,
		blockSizeX, blockSizeY, blockSizeZ, gridSizeX, gridSizeY, 0);
}

__wchar_t* DownSampleTopLeft_Parallel(
	void* deviceInputBuffer, void* deviceOutputBuffer,
	int widthSourceImage, int heightSourceImage, int strideSourceImage,
	int widthDestImage, int heightDestImage, int strideDestImage,
	double horizontalScale, double verticalScale,
	int subPixelType, int maxValue, int alphaChannelNumber, int pixelSize, int channelSize,
	int blockSizeX, int blockSizeY, int blockSizeZ, int gridSizeX, int gridSizeY, void* stream)
{


	dim3 blockDim(blockSizeX, blockSizeY, blockSizeZ); // block size = number of threads
	dim3 gridDim(gridSizeX, gridSizeY); // grid size = number of blocks

	//int numOfChannels = (pixelSize/channelSize);
	//int blockSize = (numOfChannels==1) ? 256 : 64 ; 
	//dim3 blockDim(blockSize,1,numOfChannels);
	//int numOfBlocks = ceil((widthDestImage*heightDestImage*numOfChannels)/(float)(blockSize*numOfChannels));
	//dim3 gridDim(numOfBlocks,1,1);

	switch (subPixelType)
	{
	case 1: // 8bit (0...0xff)
	{

		DownSampleTopLeftKernel<unsigned char> << <gridDim, blockDim >> >
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
			widthSourceImage, heightSourceImage, strideSourceImage,
			widthDestImage, heightDestImage, strideDestImage,
			(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
			maxValue, pixelSize, channelSize
			);
		break;

	}
	case 2: // 16bit (0...0xffff)
	{

		DownSampleTopLeftKernel<unsigned short> << <gridDim, blockDim >> >
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
			widthSourceImage, heightSourceImage, strideSourceImage,
			widthDestImage, heightDestImage, strideDestImage,
			(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
			maxValue, pixelSize, channelSize
			);
		break;

	}
	case 4: // Float (0...1)
	{

		DownSampleTopLeftKernelFloat<float> << <gridDim, blockDim >> >
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
			widthSourceImage, heightSourceImage, strideSourceImage,
			widthDestImage, heightDestImage, strideDestImage,
			(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
			pixelSize, channelSize
			);
		break;
	}
	}

	if (stream == 0)
	{
		hipError_t err = hipDeviceSynchronize();
		if (err != 0)
			return CudaErrorToErrorMessage(err);
	}
	return 0;
}



// ------------------------------------------------------------------------------------------------------------------


__wchar_t* DownSample(
		void* deviceInputBuffer, void* deviceOutputBuffer,
		int widthSourceImage, int heightSourceImage, int strideSourceImage,
		int widthDestImage, int heightDestImage, int strideDestImage,
		double horizontalScale, double verticalScale,
		int subPixelType, int maxValue, int alphaChannelNumber, int pixelSize, int channelSize,			
		int blockSizeX, int blockSizeY, int blockSizeZ, int gridSizeX, int gridSizeY)
{
	return DownSample_Parallel(deviceInputBuffer, deviceOutputBuffer, widthSourceImage, heightSourceImage, 
		strideSourceImage, widthDestImage, heightDestImage, strideDestImage, horizontalScale, verticalScale,
		subPixelType, maxValue, alphaChannelNumber, pixelSize, channelSize,			
		blockSizeX, blockSizeY, blockSizeZ, gridSizeX, gridSizeY, 0); 
}

__wchar_t* DownSample_Parallel(
		void* deviceInputBuffer, void* deviceOutputBuffer,
		int widthSourceImage, int heightSourceImage, int strideSourceImage,
		int widthDestImage, int heightDestImage, int strideDestImage,
		double horizontalScale, double verticalScale,
		int subPixelType, int maxValue, int alphaChannelNumber, int pixelSize, int channelSize,			
		int blockSizeX, int blockSizeY, int blockSizeZ, int gridSizeX, int gridSizeY, void* stream)
{


	dim3 blockDim(blockSizeX, blockSizeY, blockSizeZ); // block size = number of threads
    dim3 gridDim(gridSizeX, gridSizeY); // grid size = number of blocks
	
	//int numOfChannels = (pixelSize/channelSize);
	//int blockSize = (numOfChannels==1) ? 256 : 64 ; 
	//dim3 blockDim(blockSize,1,numOfChannels);
	//int numOfBlocks = ceil((widthDestImage*heightDestImage*numOfChannels)/(float)(blockSize*numOfChannels));
	//dim3 gridDim(numOfBlocks,1,1);

	switch (subPixelType)
	{
		case 1: // 8bit (0...0xff)
		{
		
			DownSampleKernel<unsigned char> <<<gridDim, blockDim>>>
			  ((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthSourceImage, heightSourceImage, strideSourceImage,
				widthDestImage, heightDestImage, strideDestImage,
				(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
				maxValue, pixelSize, channelSize
				);
			break;
			
		}
		case 2: // 16bit (0...0xffff)
		{
		
			DownSampleKernel<unsigned short> <<<gridDim, blockDim>>>
			  ((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthSourceImage, heightSourceImage, strideSourceImage,
				widthDestImage, heightDestImage, strideDestImage,
				(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
				maxValue, pixelSize, channelSize
				);
			break;
		
		}
		case 4: // Float (0...1)
		{
			
			DownSampleKernelFloat<float> <<<gridDim, blockDim>>>
			  ((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthSourceImage, heightSourceImage, strideSourceImage,
				widthDestImage, heightDestImage, strideDestImage,
				(float)(1.0 / horizontalScale), (float)(1.0 / verticalScale),
				pixelSize, channelSize
				);
			break;
		}
	}
	
	if(stream == 0)
	{
		hipError_t err = hipDeviceSynchronize();
		if (err != 0)
			return CudaErrorToErrorMessage(err);
	}
	return 0;
}


template< class T>
__wchar_t* RunFastDownSampleKernel(void* deviceInputBuffer, void* deviceIntegerOutputBuffer, 
								   void* deviceOutputBuffer, double maxValue, double convertValue, 
								   int numberOfChannels, int subPixelType, 
								   int widthSourceImage, int heightSourceImage, int strideSourceImage,
								   int widthDestImage, int heightDestImage, int strideIntegerDestImage, int strideDestImage,
								   double horizontalScale, double verticalScale,
								   int blockSizeSrcX, int blockSizeSrcY, int gridSizeSrcX, int gridSizeSrcY, 
								   int blockSizeDstX, int blockSizeDstY, int gridSizeDstX, int gridSizeDstY,
								   void* stream)
{



	dim3 blockDim(blockSizeSrcX, blockSizeSrcY, 1); // block size = number of threads
    dim3 gridDim(gridSizeSrcX, gridSizeSrcY); // grid size = number of blocks

	float downSampleConvertValue = (float)(convertValue * horizontalScale * verticalScale);

	switch (numberOfChannels)
	{
	case 1:
		{
			if (horizontalScale == 1)
			{
				FastDownSampleVerticalKernel<T, 1, sizeof(T) ><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage,
					 strideIntegerDestImage,
					(float)verticalScale, downSampleConvertValue);
			}
			else if (verticalScale == 1)
			{
				FastDownSampleHorizontalKernel<T, 1, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage, 
					 strideIntegerDestImage,
					(float)horizontalScale, downSampleConvertValue);
			}
			else
			{
				FastDownSampleKernel<T, 1, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage,
					strideIntegerDestImage,
					(float)horizontalScale, (float)verticalScale, downSampleConvertValue);
			}
			break;
		}
	case 3:
		{
			if (horizontalScale == 1)
			{
				FastDownSampleVerticalKernel<T, 3, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage,
					strideIntegerDestImage,
					(float)verticalScale, downSampleConvertValue);
			}
			else if (verticalScale == 1)
			{
				FastDownSampleHorizontalKernel<T, 3, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage, 
					strideIntegerDestImage,
					(float)horizontalScale, downSampleConvertValue);
			}
			else
			{
				FastDownSampleKernel<T, 3, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage,  heightSourceImage,  strideSourceImage, 
					strideIntegerDestImage,
					(float)horizontalScale, (float)verticalScale, downSampleConvertValue);
			}

			break;
		}
	case 4:
		{
			if (horizontalScale == 1)
			{
				FastDownSampleVerticalKernel<T, 4, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage, 
					 strideIntegerDestImage,
					(float)verticalScale, downSampleConvertValue);
			}
			else if (verticalScale == 1)
			{
				FastDownSampleHorizontalKernel<T, 4, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage, heightSourceImage, strideSourceImage, 
					strideIntegerDestImage,
					(float)horizontalScale, downSampleConvertValue);
			}
			else
			{
				FastDownSampleKernel<T, 4, sizeof(T)><<<gridDim, blockDim>>>
					((unsigned char*)deviceInputBuffer, (unsigned char*)deviceIntegerOutputBuffer,
					widthSourceImage,  heightSourceImage,  strideSourceImage, 
					strideIntegerDestImage,
					(float)horizontalScale, (float)verticalScale, downSampleConvertValue);
			}

			break;
		}
	default:
		return L"Unsupported number of channels";
	}

	if (subPixelType == 4)
	{
		return RunConvertFromIntegerToFloatKernel(deviceIntegerOutputBuffer, deviceOutputBuffer,
				 numberOfChannels, maxValue, convertValue,
				widthDestImage,  heightDestImage,  strideIntegerDestImage, strideDestImage,
				blockSizeDstX, blockSizeDstY, gridSizeDstX,  gridSizeDstY, stream);
	}

	return RunConvertFromIntegerKernel<T>(deviceIntegerOutputBuffer, deviceOutputBuffer,
		numberOfChannels, maxValue, convertValue,
		widthDestImage,  heightDestImage,  strideIntegerDestImage, strideDestImage,
		blockSizeDstX, blockSizeDstY, gridSizeDstX,  gridSizeDstY, stream);

}

__wchar_t* FastDownSample(void* deviceInputBuffer, void* deviceIntegerOutputBuffer, 
								   void* deviceOutputBuffer, double maxValue, double convertValue, 
								   int numberOfChannels, int subPixelType, 
								   int widthSourceImage, int heightSourceImage, int strideSourceImage,
								   int widthDestImage, int heightDestImage, int strideIntegerDestImage, int strideDestImage,
								   double horizontalScale, double verticalScale,
								   int blockSizeSrcX, int blockSizeSrcY, int gridSizeSrcX, int gridSizeSrcY, 		
								   int blockSizeDstX, int blockSizeDstY, int gridSizeDstX, int gridSizeDstY)
{
	return FastDownSample_Parallel(deviceInputBuffer, deviceIntegerOutputBuffer, deviceOutputBuffer, 
		maxValue, convertValue, numberOfChannels,  subPixelType, 
		widthSourceImage, heightSourceImage, strideSourceImage,
		widthDestImage, heightDestImage, strideIntegerDestImage, strideDestImage,  horizontalScale,  verticalScale,
		blockSizeSrcX, blockSizeSrcY, gridSizeSrcX, gridSizeSrcY, 
		blockSizeDstX, blockSizeDstY, gridSizeDstX,  gridSizeDstY, 0);
}

__wchar_t* FastDownSample_Parallel(void* deviceInputBuffer, void* deviceIntegerOutputBuffer, 
								   void* deviceOutputBuffer, double maxValue, double convertValue, 
								   int numberOfChannels, int subPixelType, 
								   int widthSourceImage, int heightSourceImage, int strideSourceImage,
								   int widthDestImage, int heightDestImage, int strideIntegerDestImage, int strideDestImage,
								   double horizontalScale, double verticalScale,
								   int blockSizeSrcX, int blockSizeSrcY, int gridSizeSrcX, int gridSizeSrcY, 
								   int blockSizeDstX, int blockSizeDstY, int gridSizeDstX, int gridSizeDstY,
								   void* stream)
{
	dim3 blockDim(blockSizeSrcX, blockSizeSrcY, 1); // block size = number of threads
    dim3 gridDim(gridSizeSrcX, gridSizeSrcY); // grid size = number of blocks

	__wchar_t* msg = 0;

	switch (subPixelType)
	{
	case 1:// 8bit (0...0xff)
		{
		msg = RunFastDownSampleKernel<unsigned char>(deviceInputBuffer, deviceIntegerOutputBuffer, 
								    deviceOutputBuffer, maxValue, convertValue, numberOfChannels, subPixelType,
								    widthSourceImage, heightSourceImage, strideSourceImage,
								    widthDestImage, heightDestImage, strideIntegerDestImage, strideDestImage,
								    horizontalScale,  verticalScale,
								    blockSizeSrcX, blockSizeSrcY, gridSizeSrcX, gridSizeSrcY, 
								    blockSizeDstX, blockSizeDstY, gridSizeDstX, gridSizeDstY, stream);

			break;
		}
	case 2:// 16bit (0...0xffff)
		{
			msg = RunFastDownSampleKernel<unsigned short>(deviceInputBuffer, deviceIntegerOutputBuffer, 
						    deviceOutputBuffer, maxValue, convertValue, numberOfChannels, subPixelType,
						    widthSourceImage, heightSourceImage, strideSourceImage,
						    widthDestImage, heightDestImage, strideIntegerDestImage, strideDestImage,
						    horizontalScale,  verticalScale,
						    blockSizeSrcX, blockSizeSrcY, gridSizeSrcX, gridSizeSrcY, 
						    blockSizeDstX, blockSizeDstY, gridSizeDstX, gridSizeDstY, stream);
			break;
		}
	case 4: // Float (0...1)
		{
			msg = RunFastDownSampleKernel<float>(deviceInputBuffer, deviceIntegerOutputBuffer, 
						    deviceOutputBuffer, maxValue, convertValue, numberOfChannels, subPixelType,
						    widthSourceImage, heightSourceImage, strideSourceImage,
						    widthDestImage, heightDestImage, strideIntegerDestImage, strideDestImage,
						    horizontalScale,  verticalScale,
						    blockSizeSrcX, blockSizeSrcY, gridSizeSrcX, gridSizeSrcY, 
						    blockSizeDstX, blockSizeDstY, gridSizeDstX, gridSizeDstY, stream);
			break;
		}
	default:
		return L"Unsupported sub pixel format";
	}
	return msg;
}