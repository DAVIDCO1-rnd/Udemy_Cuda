// Nvcc predefines the macro __HIPCC__.
// This macro can be used in sources to test whether they are currently being compiled by nvcc.
#ifndef __HIPCC__
#error Must be compiled with CUDA compiler!
#endif

#include <stdio.h>
//#include <cutil.h>
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>

#include "CudaMain.cuh"
#include "rotate_90_cpu.cuh"

#include "rotate_90_curnel_cpu.cu"

__wchar_t* rotate_90_cpu(void* deviceInputBuffer, void* deviceOutputBuffer, int subPixelType,
	int widthImage, int heightImage, int is_clockwise,
	int blockSizeX, int blockSizeY, int blockSizeZ,
	int gridSizeX, int gridSizeY)
{
	//dim3 blockDim(blockSizeX, blockSizeY, blockSizeZ); // block size = number of threads
	//dim3 gridDim(gridSizeX, gridSizeY); // grid size = number of blocks

	int pixel_size = subPixelType;
	switch (subPixelType)
	{
	case 1: // 8bit (0...0xff)
	{
		rotate_90_kernel_cpu<unsigned char>
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthImage, heightImage, pixel_size, is_clockwise,
				blockSizeX, blockSizeY, blockSizeZ,
				gridSizeX, gridSizeY);
		break;
	}
	case 2: // 16bit (0...0xffff)
	{
		rotate_90_kernel_cpu<unsigned short>
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthImage, heightImage, pixel_size, is_clockwise,
				blockSizeX, blockSizeY, blockSizeZ,
				gridSizeX, gridSizeY);
		break;
	}
	case 4: // Float (0...1)
	{
		rotate_90_kernel_cpu<float>
			((unsigned char*)deviceInputBuffer, (unsigned char*)deviceOutputBuffer,
				widthImage, heightImage, pixel_size, is_clockwise,
				blockSizeX, blockSizeY, blockSizeZ,
				gridSizeX, gridSizeY);
		break;
	}
	}

	return NULL;
}
