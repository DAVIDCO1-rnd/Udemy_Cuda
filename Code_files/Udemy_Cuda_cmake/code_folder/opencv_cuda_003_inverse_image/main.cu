#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <iostream>
#include <stdio.h>
#include <cmath>

#define USE_CUDA
//#define USE_X_DIMENSIONS_ONLY


#ifdef USE_CUDA
#include "hip/hip_runtime.h"
#include ""
static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#endif //USE_CUDA

bool read_image_from_file = true;
const int height = 3;
const int width = 5;

#ifndef USE_X_DIMENSIONS_ONLY
enum class ThreadsAndBlocksCalculations {
    Use_optimal_function = 0,
    Use_threads_as_warp_size = 1
};
#endif //USE_X_DIMENSIONS_ONLY





enum class DirectionOfRotation {
    Clockwise = 0,
    CounterClockwise = 1
};

enum class PixelType {
    UCHAR = 1,
    USHORT = 2,
    FLOAT = 4
};

void print_single_val(unsigned char* pixelData, int i, PixelType pixel_type)
{
    if (pixel_type == PixelType::UCHAR)
    {
        unsigned char current_val = pixelData[i];
        printf("0x%02x, ", current_val);
    }
    if (pixel_type == PixelType::USHORT)
    {
        unsigned char sub_pixel1 = pixelData[i + 0];
        unsigned char sub_pixel2 = pixelData[i + 1];
        unsigned short current_val = 0x100 * sub_pixel2 + sub_pixel1;
        printf("0x%04x, ", current_val);
    }
    if (pixel_type == PixelType::FLOAT)
    {
        unsigned char sub_pixel1 = pixelData[i + 0];
        unsigned char sub_pixel2 = pixelData[i + 1];
        unsigned char sub_pixel3 = pixelData[i + 2];
        unsigned char sub_pixel4 = pixelData[i + 3];
        float current_val = 4.0 * sub_pixel4 + 3.0 * sub_pixel3 + 2.0 * sub_pixel2 + 1.0 * sub_pixel1;
        printf("%f, ", current_val);
    }
}


void print_pixels_1D(std::string matrix_name, unsigned char* pixelData, int dimension1, int dimension2, PixelType pixel_type)
{
    int pixel_size = (int)pixel_type;
    printf("%s as 1D array\n", matrix_name.c_str());
    for (int i = 0; i < pixel_size * dimension1 * dimension2; i += pixel_size)
    {
        print_single_val(pixelData, i, pixel_type);
    }
    printf("\n\n");
}

void print_pixels_2D(std::string matrix_name, unsigned char* pixelData, int dimension1, int dimension2, PixelType pixel_type)
{
    int pixel_size = (int)pixel_type;
    printf("%s as 2D array\n", matrix_name.c_str());
    for (int i = 0; i < dimension1; i++)
    {
        for (int j = 0; j < pixel_size * dimension2; j += pixel_size)
        {
            int current_index = i * pixel_size * dimension2 + j;
            print_single_val(pixelData, current_index, pixel_type);
        }
        printf("\n");
    }
    printf("\n\n");
}

void print_pixels(std::string matrix_name, unsigned char* pixelData, int dimension1, int dimension2, PixelType pixel_type)
{
    print_pixels_1D(matrix_name, pixelData, dimension1, dimension2, pixel_type);
    print_pixels_2D(matrix_name, pixelData, dimension1, dimension2, pixel_type);
}

#ifdef USE_CUDA

__device__ inline int PixelOffset1D(int x, int channel, int pixelSize, int channelSize)
{
    return  x * pixelSize + channel * channelSize;
}

__device__ inline int PixelOffset(int y, int x, int channel, int stride, int pixelSize, int channelSize)
{
    return y * stride + PixelOffset1D(x, channel, pixelSize, channelSize);
}

__device__  inline bool DecodeYXC(int* y, int* x, int* c, int widthImage, int heightImage)
{
    *y = (threadIdx.y) + (blockDim.y) * (blockIdx.y);
    *x = (threadIdx.x) + (blockDim.x) * (blockIdx.x);
    *c = (threadIdx.z);

    return (*y >= 0 && *y < heightImage&&* x >= 0 && *x < widthImage);
}

template<class T> __device__  inline T* Pixel(void* buffer, int offset)
{
    return (T*)((unsigned char*)buffer + offset);
}

template<class T> __global__ void InvertImageKernel(unsigned char* inputData, unsigned char* outputData,
    T white, int alphaChannelNum, int pixelSize, int channelSize,
    int widthImage, int heightImage,
    int strideSourceImage, int strideResultImage)
{
    int row = 0;
    int column = 0;
    int channel = 0;
    if (!DecodeYXC(&row, &column, &channel, widthImage, heightImage))
        return;

    int indexDst = PixelOffset(row, column, channel, strideResultImage, pixelSize, channelSize);
    int indexSrc = PixelOffset(row, column, channel, strideSourceImage, pixelSize, channelSize);

    if (channel != alphaChannelNum) // Not alpha channel
    {
        *(Pixel<T>(outputData, indexDst)) = white - *(Pixel<T>(inputData, indexSrc)); // Inverse
    }
    else // Alpha Channel
    {
        *(Pixel<T>(outputData, indexDst)) = *(Pixel<T>(inputData, indexSrc)); // Copy 
    }

}
#endif //USE_CUDA

template <class T>
void build_image_rotated_by_90_degrees_cpu(unsigned char* inputData, unsigned char* outputData, int input_width, int input_height, int pixel_size, int direction_of_rotation)
{
    int output_width = input_height;
    int output_height = input_width;

    int i = 0;
    while (i < input_width)
    {
        int j = 0;
        while (j < input_height)
        {
            int current_index_input_data = pixel_size * (i * input_height + j);
            int current_index_output_data;

            if (direction_of_rotation == 0) //Clockwise
            {
                current_index_output_data = pixel_size * ((input_height - j - 1) * input_width + i);
            }
            else //CounterClockwise
            {
                current_index_output_data = pixel_size * (j * input_width + input_width - 1 - i);
            }
            T pixel_value = *(T*)(inputData + current_index_output_data);
            *((T*)(outputData + current_index_input_data)) = pixel_value;

            if (read_image_from_file == false)
            {
                printf("%d, ", current_index_output_data);
            }
            j++;
        }
        if (read_image_from_file == false)
        {
            printf("\n");
        }
        i++;
    }

    if (read_image_from_file == false)
    {
        printf("\n\n");
        printf("build_transposed_image_cpu\n");
        for (int i = 0; i < input_width * input_height; i++)
        {
            unsigned char current_val = outputData[i];
            printf("%d.  %d\n", i, current_val);
        }
        printf("\n\n");
    }
}

cv::Mat build_image_from_data(uchar image_data[][width], PixelType pixel_type)
{
    cv::Mat image;
    switch (pixel_type)
    {
    case PixelType::UCHAR:
        image = cv::Mat(height, width, CV_8UC1);
        for (int y = 0; y < image.rows; ++y) {
            for (int x = 0; x < image.cols; ++x) {
                image.at<uchar>(y, x) = static_cast<uchar>(image_data[y][x]);
            }
        }
        break;

    case PixelType::USHORT:
        image = cv::Mat(height, width, CV_16UC1);
        for (int y = 0; y < image.rows; ++y) {
            for (int x = 0; x < image.cols; ++x) {
                uchar current_val = image_data[y][x];
                ushort current_val_ushort = (ushort)current_val;
                ushort new_val = 0xFF00 + current_val_ushort;
                image.at<ushort>(y, x) = new_val;
            }
        }
        break;

    case PixelType::FLOAT:
        image = cv::Mat(height, width, CV_32FC1);
        for (int y = 0; y < image.rows; ++y) {
            for (int x = 0; x < image.cols; ++x) {
                uchar current_val = image_data[y][x];
                float current_val_float = (float)current_val;
                float new_val = 1.0 * current_val_float;
                image.at<float>(y, x) = new_val;
            }
        }
        break;

    }
    return image;
}

#ifdef USE_CUDA
class BlockAndGridDimensions {
public:
    dim3 blocksPerGrid;
    dim3 threadsPerBlock;
    BlockAndGridDimensions(dim3 block_sizes, dim3 grid_sizes) {
        blocksPerGrid = grid_sizes;
        threadsPerBlock = block_sizes;
    }
};

//c++ code:
BlockAndGridDimensions* CalculateBlockAndGridDimensions(int channels, int width, int height)
{
    hipDeviceProp_t  prop;
    int device_index = 0; //For now I assume there's only one GPu device
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device_index));
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    int maxBlockSize = maxThreadsPerBlock / 2;

    dim3 blockSize;
    dim3 gridSize;

    // Calculate optimal block size, depends on the number of channels in picture
    if (width * height * channels < maxBlockSize)
    {
        blockSize.x = width;
        blockSize.y = height;
    }
    else
    {
        int warpSize = prop.warpSize;
        float dWarp = warpSize / (float)channels;
        int maxSize = (int)(maxBlockSize / (float)channels);

        if (width <= maxSize)
            blockSize.x = width;
        else
        {
            float threadsX = 0.0f;
            while (threadsX < maxSize)
            {
                threadsX += dWarp;

            }
            blockSize.x = (int)threadsX;
        }
        blockSize.y = maxSize / blockSize.x;
        if (blockSize.y == 0)
        {
            blockSize.y = 1;
        }
    }

    //block size 3rd dimension is always the number of channels.
    blockSize.z = channels;

    //calculate grid size. (number of necessary blocks to cover the whole picture) 
    gridSize.x = (int)ceil((double)width / blockSize.x);
    gridSize.y = (int)ceil((double)height / blockSize.y);

    BlockAndGridDimensions* block_and_grid_dimensions = new BlockAndGridDimensions(blockSize, gridSize);
    return block_and_grid_dimensions;

    //return new BlockAndGridDimensions(
    //    blockSize,
    //    gridSize
    //);
}

//c# code:
//public static BlockAndGridDimensions CalculateBlockAndGridDimensions(int channels, int width, int height)
//{
//
//    var maxBlockSize = DeviceProperties.deviceThreadsPerBlock / 2;
//
//
//    var blockSize = new int[3];
//    var gridSize = new int[2];
//
//    // Calculate optimal block size, depends on the number of channels in picture
//    if (width * height * channels < maxBlockSize)
//    {
//        blockSize[0] = width;
//        blockSize[1] = height;
//    }
//    else
//    {
//        var dWarp = DeviceProperties.deviceWarpSize / (float)channels;
//        var maxSize = (int)(maxBlockSize / (float)channels);
//
//        if (width <= maxSize)
//            blockSize[0] = width;
//        else
//        {
//            var threadsX = 0.0f;
//            while (threadsX < maxSize)
//            {
//                threadsX += dWarp;
//
//            }
//            blockSize[0] = (int)threadsX;
//        }
//        blockSize[1] = maxSize / blockSize[0];
//        if (blockSize[1] == 0)
//        {
//            blockSize[1] = 1;
//        }
//    }
//
//    //block size 3rd dimension is always the number of channels.
//    blockSize[2] = channels;
//
//    //calculate grid size. (number of necessary blocks to cover the whole picture) 
//    gridSize[0] = (int)Math.Ceiling((double)width / blockSize[0]);
//    gridSize[1] = (int)Math.Ceiling((double)height / blockSize[1]);
//
//    return new BlockAndGridDimensions(
//        blockSize,
//        gridSize
//    );
//}
#endif //USE_CUDA

cv::Mat calc_resized_image(cv::Mat image, double scale_factor)
{

    // Calculate the new dimensions based on the scale factor
    int newWidth = static_cast<int>(image.cols * scale_factor);
    int newHeight = static_cast<int>(image.rows * scale_factor);

    // Create a new image with the scaled dimensions
    cv::Mat scaledImage;

    // Resize the image using the resize function
    cv::resize(image, scaledImage, cv::Size(newWidth, newHeight), cv::INTER_LINEAR);

    return scaledImage;
}

int main()
{
#ifndef USE_X_DIMENSIONS_ONLY
    ThreadsAndBlocksCalculations threads_and_blocks_calculations = ThreadsAndBlocksCalculations::Use_optimal_function;
#endif //USE_X_DIMENSIONS_ONLY


    //going back from this folder: ./build/code_folder/Section3.3_spotlights/
    std::string image_path = "../../../../../images/balloons.jpg";
    cv::Mat image1_uchar;
    cv::Mat image1_ushort;
    cv::Mat image1_float;
    if (read_image_from_file == true)
    {
        cv::Mat rgb_image1 = cv::imread(image_path);
        cv::cvtColor(rgb_image1, image1_uchar, cv::COLOR_BGR2GRAY);
        cv::vconcat(image1_uchar, image1_uchar, image1_uchar);
        cv::hconcat(image1_uchar, image1_uchar, image1_uchar);
        //int newWidth = 2048;
        //int newHeight = 2560;
        //cv::resize(rgb_image1, rgb_image1, cv::Size(newWidth, newHeight), cv::INTER_LINEAR);
        //cv::imwrite(image_path, rgb_image1);
        if (image1_uchar.empty())
        {
            std::cout << "Could not read the image: " << image_path << std::endl;
            return 1;
        }
        image1_uchar.convertTo(image1_ushort, CV_16UC1, 256);
        image1_uchar.convertTo(image1_float, CV_32FC1, 65536);
    }


    if (read_image_from_file == false)
    {
        //uchar image_data[height][width] = {
        //   {0x05, 0x10, 0x15, 0x20, 0x25, 0x30},
        //   {0x35, 0x40, 0x45, 0x50, 0x55, 0x60},
        //   {0x65, 0x70, 0x75, 0x80, 0x85, 0x90}
        //};

        uchar image_data[height][width] = {
           {0x00, 0x01, 0x02, 0x03, 0x04},
           {0x05, 0x06, 0x07, 0x08, 0x09},
           {0x10, 0x11, 0x12, 0x13, 0x14}
        };
        image1_uchar = build_image_from_data(image_data, PixelType::UCHAR);
        print_pixels("built-in image1_uchar", image1_uchar.data, image1_uchar.rows, image1_uchar.cols, PixelType::UCHAR);

        image1_ushort = build_image_from_data(image_data, PixelType::USHORT);
        print_pixels("built-in image1_ushort", image1_ushort.data, image1_ushort.rows, image1_ushort.cols, PixelType::USHORT);

        image1_float = build_image_from_data(image_data, PixelType::FLOAT);
        print_pixels("built-in image1_float", image1_float.data, image1_float.rows, image1_float.cols, PixelType::FLOAT);
    }



    cv::Mat image2_uchar(image1_uchar.rows, image1_uchar.cols, CV_8UC1);
    cv::Mat image2_ushort(image1_ushort.rows, image1_ushort.cols, CV_16UC1);
    cv::Mat image2_float(image1_float.rows, image1_float.cols, CV_32FC1);



    DirectionOfRotation direction_of_rotation = DirectionOfRotation::Clockwise;
#ifndef USE_CUDA
    build_image_rotated_by_90_degrees_cpu<unsigned char>(image1_uchar.data, image2_uchar.data, image1_uchar.cols, image1_uchar.rows, (int)PixelType::UCHAR, (int)direction_of_rotation);

    build_image_rotated_by_90_degrees_cpu<unsigned short>(image1_ushort.data, image2_ushort.data, image1_ushort.cols, image1_ushort.rows, (int)PixelType::USHORT, (int)direction_of_rotation);

    build_image_rotated_by_90_degrees_cpu<float>(image1_float.data, image2_float.data, image1_float.cols, image1_float.rows, (int)PixelType::FLOAT, (int)direction_of_rotation);
#endif

#ifdef USE_CUDA
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //create device_inputData1
    unsigned char* device_inputData1 = NULL;
    size_t device_inputData_bytes1 = sizeof(unsigned char) * image1_uchar.rows * image1_uchar.cols;
    HANDLE_ERROR(hipMalloc((void**)&device_inputData1, device_inputData_bytes1));
    HANDLE_ERROR(hipMemcpy(device_inputData1, image1_uchar.data, device_inputData_bytes1, hipMemcpyHostToDevice));


    //create device_inputData2
    unsigned char* device_inputData2 = NULL;
    size_t device_inputData_bytes2 = sizeof(unsigned short) * image1_ushort.rows * image1_ushort.cols;
    HANDLE_ERROR(hipMalloc((void**)&device_inputData2, device_inputData_bytes2));

    // Copy input vectors from host memory to GPU buffers.
    HANDLE_ERROR(hipMemcpy(device_inputData2, image1_ushort.data, device_inputData_bytes2, hipMemcpyHostToDevice));

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    unsigned char* device_outputData1 = NULL;
    unsigned char* device_outputData2 = NULL;
    unsigned int device_outputData_num_of_elements = image1_uchar.rows * image1_uchar.cols;
    size_t device_outputData_num_of_bytes1 = device_outputData_num_of_elements * sizeof(unsigned char);
    HANDLE_ERROR(hipMalloc((void**)&device_outputData1, device_outputData_num_of_bytes1));

    size_t device_outputData_num_of_bytes2 = device_outputData_num_of_elements * sizeof(unsigned short);
    HANDLE_ERROR(hipMalloc((void**)&device_outputData2, device_outputData_num_of_bytes2));





    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //create device_input_width
    int* device_input_width = NULL;
    size_t device_input_width_bytes = sizeof(int);
    HANDLE_ERROR(hipMalloc((void**)&device_input_width, device_input_width_bytes));
    HANDLE_ERROR(hipMemcpy(device_input_width, &(image1_uchar.cols), device_input_width_bytes, hipMemcpyHostToDevice));


    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //create device_input_height
    int* device_input_height = NULL;
    size_t device_input_height_bytes = sizeof(int);
    HANDLE_ERROR(hipMalloc((void**)&device_input_height, device_input_height_bytes));
    HANDLE_ERROR(hipMemcpy(device_input_height, &(image1_uchar.rows), device_input_height_bytes, hipMemcpyHostToDevice));



    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //create device_uchar_pixel_size
    int* device_uchar_pixel_size = NULL;
    size_t device_uchar_pixel_size_bytes = sizeof(int);
    HANDLE_ERROR(hipMalloc((void**)&device_uchar_pixel_size, device_uchar_pixel_size_bytes));
    int uchar_pixel_size = (int)PixelType::UCHAR;
    HANDLE_ERROR(hipMemcpy(device_uchar_pixel_size, &(uchar_pixel_size), device_uchar_pixel_size_bytes, hipMemcpyHostToDevice));

    //create device_ushort_pixel_size
    int* device_ushort_pixel_size = NULL;
    size_t device_ushort_pixel_size_bytes = sizeof(int);
    HANDLE_ERROR(hipMalloc((void**)&device_ushort_pixel_size, device_ushort_pixel_size_bytes));

    int ushort_pixel_size = (int)PixelType::USHORT;
    HANDLE_ERROR(hipMemcpy(device_ushort_pixel_size, &(ushort_pixel_size), device_ushort_pixel_size_bytes, hipMemcpyHostToDevice));

    int image_height = image1_uchar.rows;
    int image_width = image1_uchar.cols;
    int num_of_channels = 1;

    //int blocksPerGrid = 256;    //dridDim is two-dimensional
    //int threadsPerBlock = 256;  //blockDim is three-dimensional


    //hipDeviceProp_t  prop;
    //int device_index = 0; //For now I assume there's only one GPu device
    //HANDLE_ERROR(hipGetDeviceProperties(&prop, device_index));
    //int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    //int threadsPerBlock = std::min(image_height, maxThreadsPerBlock);
    //int blocksPerGrid = (image_height * image_width + threadsPerBlock - 1) / threadsPerBlock;

    int num_of_threads_x = 32;
    int num_of_threads_y = 32;

    int num_of_blocks_x = (image_width + num_of_threads_x - 1) / num_of_threads_x;
    int num_of_blocks_y = (image_height + num_of_threads_y - 1) / num_of_threads_y;

    dim3 blocksPerGrid;
    dim3 threadsPerBlock;

#ifdef USE_X_DIMENSIONS_ONLY
    blocksPerGrid = dim3(256, 1, 1);
    threadsPerBlock = dim3(256, 1, 1);
#else //USE_X_DIMENSIONS_ONLY
    if (threads_and_blocks_calculations == ThreadsAndBlocksCalculations::Use_optimal_function)
    {
        BlockAndGridDimensions* block_and_grid_dims = CalculateBlockAndGridDimensions(num_of_channels, image_width, image_height);
        blocksPerGrid = block_and_grid_dims->blocksPerGrid;
        threadsPerBlock = block_and_grid_dims->threadsPerBlock;
    }
    else if (threads_and_blocks_calculations == ThreadsAndBlocksCalculations::Use_threads_as_warp_size)
    {
        blocksPerGrid = dim3(num_of_blocks_x, num_of_blocks_y, 1);
        threadsPerBlock = dim3(num_of_threads_x, num_of_threads_y);
    }
#endif  //USE_X_DIMENSIONS_ONLY


    int is_clockwise = 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    unsigned char max_val_uchar = 255;
    int alphaChannelNum = -1;
    int channelSize = 1;
    int input_image_width = image1_uchar.cols;
    int input_image_height = image1_uchar.rows;
    int uchar_strideSourceImage = input_image_width * uchar_pixel_size;
    int uchar_strideResultImage = input_image_width * uchar_pixel_size;
    
    InvertImageKernel<unsigned char> << < blocksPerGrid, threadsPerBlock >> > (device_inputData1, device_outputData1,
        max_val_uchar, alphaChannelNum, uchar_pixel_size, channelSize,
        input_image_width, input_image_height,
        uchar_strideSourceImage, uchar_strideResultImage);

    channelSize = 2;
    unsigned short max_val_ushort = 65535;
    int ushort_strideSourceImage = input_image_width * ushort_pixel_size;
    int ushort_strideResultImage = input_image_width * ushort_pixel_size;
    InvertImageKernel<unsigned short> << < blocksPerGrid, threadsPerBlock >> > (device_inputData2, device_outputData2,
        max_val_ushort, alphaChannelNum, ushort_pixel_size, channelSize,
        input_image_width, input_image_height,
        ushort_strideSourceImage, ushort_strideResultImage);


    //build_image_rotated_by_90_degrees_cuda<unsigned char> << < blocksPerGrid, threadsPerBlock >> > (device_inputData1, device_outputData1, device_input_width, device_input_height, device_uchar_pixel_size, is_clockwise);
    //build_image_rotated_by_90_degrees_cuda<unsigned short> << < blocksPerGrid, threadsPerBlock >> > (device_inputData2, device_outputData2, device_input_width, device_input_height, device_ushort_pixel_size, is_clockwise);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("gpu time = milliseconds =%.8f\n", milliseconds);

    // Check for any errors launching the kernel
    HANDLE_ERROR(hipGetLastError());


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    HANDLE_ERROR(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    unsigned char* outputData1 = (unsigned char*)malloc(device_outputData_num_of_bytes1);
    HANDLE_ERROR(hipMemcpy(outputData1, device_outputData1, device_outputData_num_of_bytes1, hipMemcpyDeviceToHost));

    image2_uchar.data = outputData1;

    // Copy output vector from GPU buffer to host memory.
    unsigned char* outputData2 = (unsigned char*)malloc(device_outputData_num_of_bytes2);
    HANDLE_ERROR(hipMemcpy(outputData2, device_outputData2, device_outputData_num_of_bytes2, hipMemcpyDeviceToHost));
    image2_ushort.data = outputData2;

    HANDLE_ERROR(hipFree(device_inputData1));
    HANDLE_ERROR(hipFree(device_inputData2));
    HANDLE_ERROR(hipFree(device_outputData1));
    HANDLE_ERROR(hipFree(device_outputData2));
    HANDLE_ERROR(hipFree(device_input_width));
    HANDLE_ERROR(hipFree(device_input_height));
    HANDLE_ERROR(hipFree(device_uchar_pixel_size));
    HANDLE_ERROR(hipFree(device_ushort_pixel_size));

#endif //USE_CUDA


    if (read_image_from_file == true)
    {
        double scale_factor = 0.25;
        cv::Mat resized_image1_uchar = calc_resized_image(image1_uchar, scale_factor);
        cv::Mat resized_image2_uchar = calc_resized_image(image2_uchar, scale_factor);
        cv::Mat resized_image1_ushort = calc_resized_image(image1_ushort, scale_factor);
        cv::Mat resized_image2_ushort = calc_resized_image(image2_ushort, scale_factor);

        cv::imshow("resized_image1_uchar", resized_image1_uchar);
        cv::imshow("resized_image2_uchar", resized_image2_uchar);

        cv::imshow("resized_image1_ushort", resized_image1_ushort);
        cv::imshow("resized_image2_ushort", resized_image2_ushort);

        //cv::imshow("image1_float", image1_float);
        //cv::imshow("image2_float", image2_float);
    }
    else
    {
        print_pixels("image1_uchar", image1_uchar.data, image1_uchar.rows, image1_uchar.cols, PixelType::UCHAR);
        print_pixels("image2_uchar", image2_uchar.data, image2_uchar.rows, image2_uchar.cols, PixelType::UCHAR);

        print_pixels("image1_ushort", image1_ushort.data, image1_ushort.rows, image1_ushort.cols, PixelType::USHORT);
        print_pixels("image2_ushort", image2_ushort.data, image2_ushort.rows, image2_ushort.cols, PixelType::USHORT);

        //print_pixels("image1_float", image1_ushort.data, image1_ushort.rows, image1_ushort.cols, PixelType::FLOAT);
        //print_pixels("image2_float", image2_ushort.data, image2_ushort.rows, image2_ushort.cols, PixelType::FLOAT);
    }

    int k = cv::waitKey(0); // Wait for a keystroke in the window



    return 0;
}




//#include <iostream>
//#include <opencv2/core.hpp>
//#include <opencv2/imgcodecs.hpp>
//#include <opencv2/highgui.hpp>
//#include <hip/hip_runtime.h>
//
//// CUDA kernel code
//__global__ void multiply_by_constant(float* input, float constant, int size)
//{
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (idx < size)
//    {
//        input[idx] *= constant;
//    }
//}
//
//int main()
//{
//    // Create a sample buffer array in C++
//    int size = 9;
//    float input_buffer[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};
//
//    // Allocate memory on the GPU
//    float* d_input_buffer;
//    hipMalloc((void**)&d_input_buffer, size * sizeof(float));
//
//    // Copy the input buffer from the CPU to the GPU
//    hipMemcpy(d_input_buffer, input_buffer, size * sizeof(float), hipMemcpyHostToDevice);
//
//    // Define the block and grid dimensions for CUDA execution
//    int block_size = 256;
//    int num_blocks = (size + block_size - 1) / block_size;
//
//    // Execute the CUDA kernel
//    multiply_by_constant<<<num_blocks, block_size>>>(d_input_buffer, 2.0f, size);
//
//    // Copy the result back from the GPU to the CPU
//    float output_buffer[size];
//    hipMemcpy(output_buffer, d_input_buffer, size * sizeof(float), hipMemcpyDeviceToHost);
//
//    // Clean up memory on the GPU
//    hipFree(d_input_buffer);
//
//    // Show the result using OpenCV (just as an example)
//    cv::Mat result = cv::Mat(1, size, CV_32F, output_buffer);
//    std::cout << "Input Buffer: " << cv::Mat(1, size, CV_32F, input_buffer) << std::endl;
//    std::cout << "Output Buffer: " << result << std::endl;
//
//    return 0;
//}