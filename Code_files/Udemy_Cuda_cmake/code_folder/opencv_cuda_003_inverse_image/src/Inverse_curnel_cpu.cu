#ifndef __HIPCC__
#error Must be compiled with CUDA compiler!
#endif

#pragma once

#include "Utils_device_cpu.cu"

//defines a global function called from the host (CPU) excuted on the device (GPU)
template<class T> void InvertImageKernel_cpu(unsigned char* inputData, unsigned char* outputData,
													T white, int alphaChannelNum, int pixelSize, int channelSize,
													int widthImage, int heightImage, 
													int strideSourceImage, int strideResultImage,
													int blockSizeX, int blockSizeY, int blockSizeZ,
													int gridSizeX, int gridSizeY)
{
	int block_Dim_x = blockSizeX;
	int block_Dim_y = blockSizeY;

	for (int thread_Idx_y = 0; thread_Idx_y < blockSizeY; thread_Idx_y++)
	{
		for (int thread_Idx_x = 0; thread_Idx_x < blockSizeX; thread_Idx_x++)
		{
			for (int thread_Idx_z = 0; thread_Idx_z < blockSizeZ; thread_Idx_z++)
			{
				for (int block_Idx_x = 0; block_Idx_x < gridSizeX; block_Idx_x++)
				{
					for (int block_Idx_y = 0; block_Idx_y < gridSizeY; block_Idx_y++)
					{
						int row = 0;
						int column = 0;
						int channel = 0;
						if (!DecodeYXC_cpu(&row, &column, &channel, widthImage, heightImage, thread_Idx_x, thread_Idx_y, thread_Idx_z, block_Idx_x, block_Idx_y, block_Dim_x, block_Dim_y))
							break;

						int indexDst = PixelOffset_cpu(row, column, channel, strideResultImage, pixelSize, channelSize);
						int indexSrc = PixelOffset_cpu(row, column, channel, strideSourceImage, pixelSize, channelSize);

						T current_val = *(Pixel_cpu<T>(inputData, indexSrc));
						if (channel != alphaChannelNum) // Not alpha channel
						{	
							T new_val = white - current_val;
							*(Pixel_cpu<T>(outputData, indexDst)) = new_val; // Inverse
						}
						else // Alpha Channel
						{
							*(Pixel_cpu<T>(outputData, indexDst)) = current_val; // Copy 
						}
					}
				}
			}
		}
	}
}
