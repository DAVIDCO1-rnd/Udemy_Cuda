#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <iostream>


int main()
{
    //going back from this folder: ./build/code_folder/Section3.3_spotlights/
    std::string image_path = "../../../code_folder/opencv_cuda/images/00013.jpg";
    cv::Mat img = cv::imread(image_path);

    if (img.empty())
    {
        std::cout << "Could not read the image: " << image_path << std::endl;
        return 1;
    }

    cv::imshow("Display window", img);
    int k = cv::waitKey(0); // Wait for a keystroke in the window
    if (k == 's')
    {
        cv::imwrite("starry_night.png", img);
    }


    return 0;
}


//#include <iostream>
//#include <opencv2/core.hpp>
//#include <opencv2/imgcodecs.hpp>
//#include <opencv2/highgui.hpp>
//#include <hip/hip_runtime.h>
//
//// CUDA kernel code
//__global__ void multiply_by_constant(float* input, float constant, int size)
//{
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (idx < size)
//    {
//        input[idx] *= constant;
//    }
//}
//
//int main()
//{
//    // Create a sample buffer array in C++
//    int size = 9;
//    float input_buffer[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};
//
//    // Allocate memory on the GPU
//    float* d_input_buffer;
//    hipMalloc((void**)&d_input_buffer, size * sizeof(float));
//
//    // Copy the input buffer from the CPU to the GPU
//    hipMemcpy(d_input_buffer, input_buffer, size * sizeof(float), hipMemcpyHostToDevice);
//
//    // Define the block and grid dimensions for CUDA execution
//    int block_size = 256;
//    int num_blocks = (size + block_size - 1) / block_size;
//
//    // Execute the CUDA kernel
//    multiply_by_constant<<<num_blocks, block_size>>>(d_input_buffer, 2.0f, size);
//
//    // Copy the result back from the GPU to the CPU
//    float output_buffer[size];
//    hipMemcpy(output_buffer, d_input_buffer, size * sizeof(float), hipMemcpyDeviceToHost);
//
//    // Clean up memory on the GPU
//    hipFree(d_input_buffer);
//
//    // Show the result using OpenCV (just as an example)
//    cv::Mat result = cv::Mat(1, size, CV_32F, output_buffer);
//    std::cout << "Input Buffer: " << cv::Mat(1, size, CV_32F, input_buffer) << std::endl;
//    std::cout << "Output Buffer: " << result << std::endl;
//
//    return 0;
//}