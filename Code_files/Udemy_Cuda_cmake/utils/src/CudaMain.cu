// Nvcc predefines the macro __HIPCC__.
// This macro can be used in sources to test whether they are currently being compiled by nvcc.
#ifndef __HIPCC__
#error Must be compiled with CUDA compiler!
#endif


#include <hip/hip_runtime_api.h>
#include "CudaMain.cuh"

__wchar_t* CudaErrorToErrorMessage(hipError_t err)
{
	switch (err)
	{
	case hipSuccess:
		return NULL;
	case hipErrorMissingConfiguration:
		return L"Missing configuration";
	case hipErrorOutOfMemory:
		return L"Memory allocation error";
	case hipErrorNotInitialized:
		return L"Initialization error";
	case hipErrorLaunchFailure:
		return L"Launch failure";
	case hipErrorPriorLaunchFailure:
		return L"Prior launch failure";
	case hipErrorLaunchTimeOut:
		return L"Launch timeout";
	case hipErrorLaunchOutOfResources:
		return L"Launch out of resources";
	case hipErrorInvalidDeviceFunction:
		return L"Invaild device function";
	case hipErrorInvalidConfiguration:
		return L"Invalid configration";
	case hipErrorInvalidDevice:
		return L"Invalid device";
	case hipErrorInvalidValue:
		return L"Invalid value";
	case hipErrorInvalidPitchValue:
		return L"Invalid pitch value";
	case hipErrorInvalidSymbol:
		return L"Invalid symbol";
	case hipErrorMapFailed:
		return L"Map of buffer object failed";
	case hipErrorUnmapFailed:
		return L"Unmap of buffer object failed";
	case cudaErrorInvalidHostPointer:
		return L"Invalid (host) pointer";
	case hipErrorInvalidDevicePointer:
		return L"Invalid GPU (device) pointer";
	case hipErrorInvalidTexture:
		return L"Invalid texture";
	case cudaErrorInvalidTextureBinding:
		return L"Invalid texture binding";
	case hipErrorInvalidChannelDescriptor:
		return L"Invalid channel descriptor";
	case hipErrorInvalidMemcpyDirection:
		return L"Invalid memory copy direction";
	case cudaErrorAddressOfConstant:
		return L"Address of constant (!?)";
	case cudaErrorTextureFetchFailed:
		return L"Texture fetch failed";
	case cudaErrorTextureNotBound:
		return L"Texture not bound";
	case cudaErrorSynchronizationError:
		return L"Synchronization error";
	case cudaErrorInvalidFilterSetting:
		return L"Invalid filter setting";
	case cudaErrorInvalidNormSetting:
		return L"Invalid normal setting";
	case cudaErrorMixedDeviceExecution:
		return L"Mixed device execution";
	case hipErrorDeinitialized:
		return L"Cudart unloading (!?)";
	case hipErrorUnknown:
		return L"Unknown error";
	case cudaErrorNotYetImplemented:
		return L"Not implemented yet";
	case cudaErrorMemoryValueTooLarge:
		return L"Memory value is too large";
	case hipErrorInvalidHandle:
		return L"Invalid resource handle";
	case hipErrorNotReady:
		return L"Not ready... come back later";
	case hipErrorSetOnActiveProcess:
		return L"hipErrorSetOnActiveProcess";		
	case hipErrorNoDevice:
		return L"hipErrorNoDevice";		
	case cudaErrorDevicesUnavailable:
		return L"cudaErrorDevicesUnavailable";
	case cudaErrorStartupFailure:
		return L"Start up failure";
	case cudaErrorApiFailureBase:
		return L"API failure";
	default:
		return L"Other error";
	}
}

